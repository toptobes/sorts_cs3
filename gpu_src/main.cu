#include "hip/hip_runtime.h"
#include "main.cuh"

#define PRINT_STATS

int main()
{
    sorting_test();
}

void sorting_test()
{
    std::srand(time(NULL));

    // This weird stuff makes numbers printed in stdout using cout
    // go from '10000000.123456' to '10,000,000.1234' (somehow)
    // I don't get it myself, but whatever. It works, so it works.
    std::locale comma_locale(std::locale(), new comma_numpunct());
    std::cout.imbue(comma_locale);
    std::cout << std::fixed << std::showpoint;
    std::cout << std::setprecision(4);

    printGpuMemoryUsageStatistics();

    START_TIMER(overall)

    // Initialize 'host' array, aka the array that can be seen by the CPU
    // Prefixed with 'h_' to indicate it's on the host
    // Code run on the CPU can't access 'device memory', AKA memory on the GPU
    // and vice versa.
    // It needs to be copied back and forth via the PCI-e bus
    int* h_nums = new int[N];

    // Initialize device arrays
    // Prefixed with 'd_' because you can probably guess why
    // Needs to be allocated with hipMalloc so it's allocated on the GPU
    // and not the CPU
    // TODO: Maybe use hipMallocManaged for d_nums & test speed differences
    int* d_nums, *d_nums_alt;
    hipMalloc(&d_nums, N_BYTES);
    hipMalloc(&d_nums_alt, N_BYTES);

    START_TIMER(arrayinit)

    // Initialize states for the random number generator
    // Need to have one for each thread that will be used
    uint64_t seed = (SEED) ? SEED : genRandomSeed();

    // Calculates the number of thread blocks that need to be generated depending
    // on how large the thread block can be
    // For example, 10 blocks for 10*1024 threads.
    int num_blocks = 1 + ((N / BATCH_SIZE) + BLOCK_SIZE - 1) / BLOCK_SIZE;

    // Runs setupNums on the GPU
    // <<< Number of threads blocks, number of threads per block>>>
    // The dimensions can be 3d, but keeping it 1d for this
    // The function is called once in thread that is run
    setupNums<<< num_blocks, BLOCK_SIZE >>>(seed, d_nums);
    hipDeviceSynchronize();

    STOP_TIMER(arrayinit)
    START_TIMER(sortingwithsetup)

    // Double buffer to save on required device storage space, ~O(N + P) vs ~O(2N + P)
    hipcub::DoubleBuffer<int> d_nums_buffer(d_nums, d_nums_alt);

    // Initialize auxiliary space needed for the sort
    void* d_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;

    // Figure out how much aux. space needed and allocate it
    // Just running the sort with d_temp_storage being null will calculate the number of
    // bytes needed and put it in temp_storage_bytes for us
    hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_nums_buffer, N);

    // We can then manually allocate the auxiliary space on the device ourselves
    hipMalloc(&d_temp_storage, temp_storage_bytes);

    // Waits for the kernal function to finish
    // Otherwise it runs asynchronously to the CPU code until another GPU
    // function is called
    // I normally wouldn't need this because hipcub::DeviceRadixSort::SortKeys will wait
    // for hipMalloc to finish, butttt I need hipMalloc to finish BEFORE timer3 starts
    hipDeviceSynchronize();

    // Finally sort array
    START_TIMER(sorting)
    hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_nums_buffer, N);

    printGpuMemoryUsageStatistics();

    // Stops the above code so timer3 can stop right after the sort is done, and not before or after
    hipDeviceSynchronize();
    STOP_TIMER(sorting)

    // Copy sorted device array back to host
    // since we can't see the device memory from the host
    // without first copying it over
    hipMemcpy(h_nums, d_nums_buffer.Current(), N_BYTES, hipMemcpyDeviceToHost);
    STOP_TIMER(sortingwithsetup)

    // Frees the device memory we don't need anymore
    hipFree(d_nums_buffer.Alternate());
    hipFree(d_temp_storage);

    STOP_TIMER(overall)

    puts("\n-----------------------------------------------------------------");
    std::cout << "- " << N << " elements sorted in:" << std::endl;
    printf(
        "- \n"
        "- JUST sorting:                              %5lldms\n"
        "- Sorting + aux. allocations & such:         %5lldms\n"
        "- \n"
        "- Overall time (not including verification): %5lldms\n"
        "- Array initialization:                      %5lldms\n"
        "-----------------------------------------------------------------\n",
        timer_sorting, timer_sortingwithsetup, timer_overall, timer_arrayinit
    );

    // Some checks and verification
    const int NUMS_TO_PRINT_FOR_VERIFICATION = 5;

    for (int i = 0; i < NUMS_TO_PRINT_FOR_VERIFICATION; i++)
    {
        if (i == 0)
        {
            puts("");
        }

        printf("| %d <- [%d]\n", h_nums[i], i);
    }

    for (int i = N - NUMS_TO_PRINT_FOR_VERIFICATION; i < N; i++)
    {
        if (i == 0)
        {
            puts("");
        }

        printf("| %d <- [%d]\n", h_nums[i], i);
    }

    bool *is_sorted = nullptr;
    hipMallocManaged(&is_sorted, sizeof(bool));

    isSorted<<< num_blocks, BLOCK_SIZE >>>(d_nums_buffer.Current(), is_sorted);
    hipDeviceSynchronize();

    if (*is_sorted)
    {
        std::cout << "\nIt wprked!\n";
    }
    else
    {
        std::cout << "\nflip.\n";
    }

#ifdef PRINT_STATS
    // TODO: Parallelize
    START_TIMER(stats)
    auto stats = intSummaryStatistics(h_nums, N);
    STOP_TIMER(stats)

    auto printStat = [](std::string msg, auto stat) {
        if (stat == (int64_t)stat)
        {
            std::cout << msg            << std::setw(22) << std::right << (int64_t)stat << std::endl;
        }
        else
        {
            std::cout << msg << "     " << std::setw(22) << std::right << stat          << std::endl;
        }
    };

    puts("\n-----------------------------------------------------------------");
    std::cout << "- Random stats (time taken: " << timer_stats << "ms)" << std::endl;
         puts( "-");
    printStat( "- Sum:        ", stats.sum        );
    printStat( "- Mean:       ", stats.mean       );
    printStat( "- Median:     ", stats.median     );
    printStat( "- Mode:       ", stats.mode       );
    printStat( "- Mode cnt:   ", stats.mode_count );
    printStat( "- Range:      ", stats.range      );
    printStat( "  - max:      ", UINT_MAX         );
    printStat( "- Std. dev:   ", stats.stdev      );
    puts("-----------------------------------------------------------------");
#endif

    hipFree(d_nums_buffer.Current());
    delete h_nums;
}

// Sets each index in the array 'xs' to a random number between INT_MIN to INT_MAX
// prefixed with __global__ to indicate it runs on the *device* (the GPU)
__global__
void setupNums(uint64_t seed, int* xs)
{
    // Gets the index of the current thread
    // I'm oversimplifying here, but threads are grouped into 3D blocks of threads,
    // but here I chose to just use the x dimension for simplicity

    // Block 1          Block 2          Block 3                              |
    // ---------------  ---------------  ---------------    this is a thread: |
    // | 0 1 2 3 4 5 |  | 0 1 2 3 4 5 |  | 0 1 2 3 4 5 |                      v
    // | | | | | | | |  | | | | | | | |  | | | | | | | |
    // | v v v v v v |  | v v v v v v |  | v v v v v v |    each block has width
    // ---------------  ---------------  ---------------    of 6 threads here

    // So, thread #0 of block #2 would be 0 + (2 * 6), which is index 12
    uint64_t idx = threadIdx.x + blockDim.x * blockIdx.x;

    // Like many C-like or functional constructs, the data is separate from the things that manipulate it.
    // Because of this, we need to manually initialize it and pass it into the RNG ourselves.
    hiprandState rand_state;
    hiprand_init(seed + idx, idx, 0, &rand_state);

    // I'm dividing the setting of each index into batches of 20000 indices per thread
    // to save on memory space and threads
    for (int i = 0; i < BATCH_SIZE; i++)
    {
        // Checking to make sure that the array index is within bounds
        // It might be out of bounds if we have extra threads
        if (i + idx * BATCH_SIZE < N)
        {
            // hiprand_uniform_double generates a uniform distribution of doubles
            // so like each number is basically equally likely to be picked
           xs[i + idx * BATCH_SIZE] = (int)(INT_MAX * hiprand_uniform_double(&rand_state));
        }
    }
}

// Just a utility function for testing purposes
// Code should be self-explanatory
void printGpuMemoryUsageStatistics()
{
    static bool have_printed_newline = false;

    if (!have_printed_newline)
    {
        puts("");
        have_printed_newline = true;
    }

    size_t free_db;
    size_t total_db;

    hipMemGetInfo(&free_db, &total_db);

    size_t used_db = total_db - free_db;

    printf(
//        "GPU memory usage: used = %2f MB, free = %2f MB, total = %f MB\n",
        "GPU memory usage: used = %2f MB\n",
        used_db  / (1024.0 * 1024.0)
//        free_db  / (1024.0 * 1024.0),
//        total_db / (1024.0 * 1024.0)
    );
}

// just read the function name and figure it out yourself
__global__
void isSorted(const int* xs, bool *is_sorted)
{
    int64_t idx = threadIdx.x + blockDim.x * blockIdx.x;

    if (idx == 0)
    {
        *is_sorted = true;
    }

    __syncthreads();

    int64_t real_index;

    for (int i = 1; i < BATCH_SIZE && *is_sorted; i++)
    {
        real_index = i + idx * BATCH_SIZE;

        if (real_index < N && xs[real_index - 1] > xs[real_index])
        {
            *is_sorted = false;
        }
    }
}

// Utility method to find the above statistics in the struct for the
// sorted array. Does not work with an unsorted array. I just made it
// for fun because why not.
IntSummaryStatistics intSummaryStatistics(const int *arr, size_t size)
{
    IntSummaryStatistics stats = {};

    int max_mode_count = 0;
    int curr_mode_count = 0;

    uint64_t sum_squared = 0;

    for (int i = 0; i < size; i++)
    {
        stats.sum += arr[i];

        double delta = arr[i] - stats.mean;

        stats.mean += delta / (i + 1);

        sum_squared += delta * (arr[i] - stats.mean);

        if (i < 1) continue;

        curr_mode_count++;

        if (arr[i] != arr[i-1] || i == size - 1)
        {
            if (curr_mode_count > max_mode_count)
            {
                stats.mode = arr[i-1];              // Covers edge case when most freq is last, and makes sure the last
                stats.mode_count = curr_mode_count + (arr[i] == arr[i-1] && i == size - 1); // item isn't different too
            }
            max_mode_count = std::max(curr_mode_count, max_mode_count);
            curr_mode_count = 0;
        }
    }

    double variance = sum_squared / (double) size;

    stats.stdev = sqrt(variance);

    if (size % 2 == 0)
    {
        stats.median = (arr[(size-1)/2] + arr[size/2]) / 2.0;
    }
    else
    {
        stats.median = arr[size/2];
    }

    // Range is range.
    stats.range = (int64_t) arr[size-1] - arr[0];

    return stats;
}

// Generates a 64-bit random number but only fills in the first
// 32 bits because I'm lazy, and it's enough for now
uint64_t genRandomSeed()
{
    uint8_t  r1 = std::rand() & 255;
    uint16_t r2 = std::rand() & 255;
    uint32_t r3 = std::rand() & 255;
    uint32_t r4 = std::rand() & 255;
    return r4 << 24 | r3 << 16 | r2 << 8 | r1 << 0;
}

// It works
char comma_numpunct::do_thousands_sep() const
{
    return ',';
}

// Somehow. Don't ask me.
std::string comma_numpunct::do_grouping() const
{
    return "\03";
}
